
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAX_WORDS 67108864
#define NUMBER_OF_TESTS 100

// Device initialization
void init_gpu(hipDeviceProp_t *deviceProp){
	// Detect GPU
	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	if (error_id != hipSuccess){
		printf("Error: hipGetDeviceCount returns %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}
	if (deviceCount == 0){
		printf("No CUDA device found\n");
		exit(EXIT_FAILURE);
	}
	// Use the first device found
	hipSetDevice(0);
	hipGetDeviceProperties(deviceProp, 0);
}

int main(int argc, char **argv){
  // Host buffer
  double *buf;
  // Device buffer
  double *devbuf;
  // Measure execution time
	struct timespec starttime, endtime;
  double elapsedtime;
  clock_t starttic, endtic, tics;
  // Counters
  size_t i, w, t;

	// GPU identification and initialization
  hipDeviceProp_t deviceProp;
	init_gpu(&deviceProp);

  printf("Iteration\tWords\tBytes\tTicks\tMbit/sec\n");

  for ( i = 0; pow(2, i) <= MAX_WORDS; i++ ){
    // Amount of words (word = double)
    w = pow(2, i);

    // Allocate host buffer
    hipHostMalloc(&buf, w * sizeof(double), hipHostMallocDefault);

    // Allocate device buffer
    hipMalloc(&devbuf, w * sizeof(double));

    elapsedtime = 0;
    tics = 0;
    for ( t = 0; t < NUMBER_OF_TESTS; t++ ){
      clock_gettime(CLOCK_MONOTONIC, &starttime);
      starttic = clock();
      hipMemcpy(devbuf, buf, w * sizeof(double), hipMemcpyHostToDevice);
      clock_gettime(CLOCK_MONOTONIC, &endtime);
      endtic = clock();
      elapsedtime += ( endtime.tv_sec - starttime.tv_sec ) + ( endtime.tv_nsec - starttime.tv_nsec ) / 1e9;
      tics += endtic - starttic;
    }

    // Elapsed time 
    elapsedtime /= NUMBER_OF_TESTS;
    tics /= NUMBER_OF_TESTS;
    // Mbits / sec
    double bandwidth;
    bandwidth = w * sizeof(double) * 1.0e-6 * 8 / elapsedtime;
    printf("%ld\t%ld\t%ld\t%ld\t%ld\n", i, w, w * sizeof(double), (size_t)tics, (size_t)bandwidth);
    elapsedtime = 0;

    hipFree(devbuf);
    hipHostFree(buf);
  }

  exit(EXIT_SUCCESS);
}
